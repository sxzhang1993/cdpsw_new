#include "hip/hip_runtime.h"
#include "../headers/book.h"
#include "stdlib.h"
#include <hip/hip_runtime.h>

#include <chrono>
#include <fstream>
#include <ios>
#include <iostream>
#include <stdio.h>
#include <string>
#include <vector>
#define delta(X,Y) ((X == Y) ? 1 : 0)
#define ceilDiv(X, Y) (((X) + (Y) - 1) / (Y))
#define max2(A,B) ((A) > (B) ? (A) : (B))
#define max4(A,B,C,D) (max2((A) , max2( (B) , max2( (C) , (D)))))
#include "unixtimer.h"

__device__ int g_uids = 0;

char * readFragment(FILE * file, size_t size){

	char * str;
	int c;
	size_t length = 0;

	str = (char *) realloc(NULL, sizeof(char) * size);
	if(!str)return str;
	while( (c = fgetc(file)) != EOF && c != '\n'){
		str[length++] = c;
		if(length == size){
			str = (char*) realloc(str, sizeof(char) * (size+=size));
			if(!str)return str;
		}
	}
	str[length++] = '\0';

	return (char *)realloc(str, sizeof(char) * length);


}



typedef struct kernelData{
	int start;
	int end;
	int windowLength;
	int xlength;
	int * matrix;
}kData;

typedef struct resultData{
	char * cigar;
	int score;
	int location;

}result;


__device__ char * my_strcpy(char *dest, const char *src){
	int i = 0;
	do {
		dest[i] = src[i];}
	while (src[i++] != 0);
	return dest;
}


__device__ char * my_strcat(char *dest, const char *src){
	int i = 0;
	while (dest[i] != 0) i++;
	my_strcpy(dest+i, src);
	return dest;
}

char * compressCigar(char * uncompressedCigar){

	int length = strlen(uncompressedCigar);
	char * compressedCigar = (char*) calloc(sizeof(char), length);
	int start = length-1;

	while(start > 0){

		if(uncompressedCigar[start] == uncompressedCigar[start-1]){
			int count = 1;

			while(uncompressedCigar[start] == uncompressedCigar[start-1]){

				count++;
				start--;

			}

			char buf[15];
			sprintf(buf, "%d%c", count, uncompressedCigar[start]);
			strcat(compressedCigar, buf);

		}else{
			char * buf = (char*) calloc(sizeof(char), 1);
			*buf = uncompressedCigar[start];
			strcat(compressedCigar, buf);
			free(buf);	
		}


		start--;
	}


	return compressedCigar;
}

/*__global__ void cdprun(int const iSize,int iDepth, char * x, char * y, kData* data, result * results) {
    
//   int tid = threadIdx.x；

	char * c_x = x;
	char * c_y = y;
//	 kData* host_data = (kData*) malloc(sizeof(kData) * nWindows);
        kData* device_data = data;
	 result * device_results = results;


	     
   // printf("Recursion=%d: Hello World from thread %d block %d\n",iDepth,threadIdx.x,blockIdx.x);
    // condition to stop recursive execution
    if (iSize == 1) return;
    // reduce block size to half
    int nthreads = iSize>>1;
    // thread 0 launches child grid recursively
    if(threadIdx.x == 0 && nthreads > 0) {
        //nestedHelloWorld<<<1, nthreads>>>(nthreads,++iDepth);
	alignKernel<<<1, nthreads>>>(c_x, c_y, device_data, device_results);
       // printf("-------> nested execution depth: %d\n",iDepth);
    }
}*/

/*__global__ void cdp_kernel(int max_depth, int depth, int thread,
                           int parent_uid) {
  // We create a unique ID per block. Thread 0 does that and shares the value
  // with the other threads.
  __shared__ int s_uid;

  if (threadIdx.x == 0) {
    s_uid = atomicAdd(&g_uids, 1);
    //atomicAdd() reads a word at some address in global or shared memory, adds a number to it, and writes the result back to the same address. Atomic functions can only be used in device functions.
  }

  __syncthreads();

  // We print the ID of the block and information about its parent.
 // print_info(depth, thread, s_uid, parent_uid);

  // We launch new blocks if we haven't reached the max_depth yet.
  if (++depth >= max_depth) {
    return;
  }

  cdp_kernel<<<gridDim.x, blockDim.x>>>(max_depth, depth, threadIdx.x, s_uid);
  //number of blocks in a grid
  //number of threads in a block
  //A group of threads is called a CUDA block. CUDA blocks are grouped into a grid. A kernel is executed as a grid of blocks of threads
}*/
__global__ void alignKernel(char * x, char * y, kData* data, result * results){



	int id = blockIdx.x;
	int start = data[id].start;
	int end = data[id].end;

	int length = end-start;
	int n = data[id].xlength;
	int * device_matrix = data[id].matrix;	
	int max = 0, innerX, innerY;

	for(int i = 0; i <= n; i++){
		device_matrix[i * length + 0] = 0; 
	}
	for(int j = 0; j <= length; j++){
		device_matrix[0 * length + j] = 0;
	}

	for(int i = 1; i <= n; i++){
		for(int j = 1; j <= length; j++){

			int val = max4(0, device_matrix[(i-1)*length+j] -1, device_matrix[i*length+ (j-1)] -1, device_matrix[((i-1)*length+(j-1))] + delta(x[i-1], y[start+j-1]));
			device_matrix[i*length+j] = val; 
			if(val > max){
				max = val;
				innerX = i;
				innerY = j;
			}
		}
	}
	int xCord = innerX, yCord = innerY;
	
	result myResults = results[id];
	while(device_matrix[xCord* length + yCord] > 0 && (yCord > 0 && yCord > 0)){
		if(device_matrix[xCord* length +yCord] == device_matrix[(xCord-1)* length +(yCord-1)] + delta(x[xCord-1], y[start + yCord-1])){

			my_strcat(myResults.cigar, "M");
			xCord--;
			yCord--;

		}else{
			if(device_matrix[xCord* length + yCord] == device_matrix[(xCord-1) * length + yCord] - 1){
				my_strcat(myResults.cigar, "I");
				xCord--;
			}else if(device_matrix[xCord * length + yCord] == device_matrix[xCord * length +(yCord-1)] -1){
				my_strcat(myResults.cigar, "D");
				yCord--;
			}	
		}
	}
	myResults.location = yCord + start + 1;
	myResults.score = max;
	results[id] = myResults;
	return;
}

__global__ void cdprun(int const iSize,int iDepth, char * x, char * y, kData* data, result * results) {

//   int tid = threadIdx.x；

        char * c_x = x;
        char * c_y = y;
//       kData* host_data = (kData*) malloc(sizeof(kData) * nWindows);
        kData* device_data = data;
         result * device_results = results;



   // printf("Recursion=%d: Hello World from thread %d block %d\n",iDepth,threadIdx.x,blockIdx.x);
    // condition to stop recursive execution
    if (iSize == 1) return;
    // reduce block size to half
    int nthreads = iSize>>1;
    // thread 0 launches child grid recursively
    if(threadIdx.x == 0 && nthreads > 0) {
        //nestedHelloWorld<<<1, nthreads>>>(nthreads,++iDepth);
        alignKernel<<<1, nthreads>>>(c_x, c_y, device_data, device_results);
       // printf("-------> nested execution depth: %d\n",iDepth);
    }
}


void print_usage(char * cmd){


	fprintf(stderr, "Usage: %s ", cmd);
	fprintf(stderr, "[-threads] ");
	fprintf(stderr, "[-overlap] ");
	fprintf(stderr, "[-largefile] ");
	fprintf(stderr, "[-smallfile] ");
	fprintf(stderr, "[-windowsize] \n");

}

int main(int argc, char * argv[]){

	FILE * xFile = stdin, * yFile = stdin;

	int numThreads = 16, windowSize = 0, overlap = 0;

	for(int i = 1; i < argc; i++){

		if(!strncmp(argv[i], "-t", strlen("-t"))){
			int userInput = atoi(argv[++i]);
			if(userInput < 16){
				printf("Invalid thread size entered. Using default thread number: %d\n", numThreads);

			}else{

				numThreads = userInput;	

			}

		}else if(!strncmp(argv[i], "-o", strlen("-o"))){

			overlap = atoi(argv[++i]);

		}else if(!strncmp(argv[i], "-w", strlen("-w"))){

			windowSize = atoi(argv[++i]);

		}else if(!strncmp(argv[i], "-s", strlen("-s"))){

			xFile =fopen(argv[++i], "r+");

		}else if(!strncmp(argv[i], "-l", strlen("-l"))){

			yFile = fopen(argv[++i], "r++");

		}else{
			print_usage(argv[0]);	
		}
	}

	if(xFile == stdin)
		printf("Please enter the smaller fragment: ");

	char * xFragment;
	xFragment = readFragment(xFile, 256);

	if(yFile == stdin)
		printf("Please enter the larger fragment: ");
	char * yFragment;
	yFragment = readFragment(yFile, 2048);

	int lenX = strlen(xFragment), lenY = strlen(yFragment);

	if(overlap == 0)	
		overlap = lenX;

	if(windowSize == 0)
		windowSize = lenX * 3;

	int nWindows = ceilDiv(lenY, windowSize);

	char * x, *y;
	HANDLE_ERROR(hipMalloc((void**) &x, sizeof(char) * lenX));
	HANDLE_ERROR(hipMalloc((void**) &y, sizeof(char) * lenY));
	HANDLE_ERROR(hipMemcpy(x, xFragment, sizeof(char) * lenX, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(y, yFragment, sizeof(char) * lenY, hipMemcpyHostToDevice));

	kData* host_data = (kData*) malloc(sizeof(kData) * nWindows);
	kData* device_data;
	HANDLE_ERROR(hipMalloc((void**) &device_data, sizeof(kData) * nWindows));
	//We now have our initialized data;

	result * host_results = (result *) malloc(sizeof(result) * nWindows);
	result * device_results;
	HANDLE_ERROR(hipMalloc((void**) &device_results, sizeof(result) * nWindows));
	//Initialized result structs

	char * cigs[nWindows];

	for(int i = 0; i < nWindows; i++){
		int start = 0;
		if(i == 0)
			start = 0;
		else
			start = host_data[i-1].start - overlap + windowSize;

		host_data[i].start = start;
		int end = start + windowSize;
		end = (end > lenY ? lenY : end);
		host_data[i].end = end;	
		host_data[i].xlength = lenX;
		host_data[i].windowLength = windowSize;
		cigs[i] = (char *) malloc(sizeof(char) * lenX * 2);
		HANDLE_ERROR(hipMalloc(&(host_results[i].cigar), sizeof(char) * lenX * 2));
		HANDLE_ERROR(hipMalloc(&(host_data[i].matrix), sizeof(int) * (lenX +1) * (windowSize + 1)));

	}
	HANDLE_ERROR(hipMemcpy(device_data, host_data, sizeof(kData) * nWindows, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(device_results, host_results, sizeof(result) * nWindows, hipMemcpyHostToDevice));


	int NITER = 10;
	int blocksize = 8;
	 dim3 block (blocksize, 1);
	//start_timer();
	for(int i = 0; i < NITER; i++){
	//	alignKernel<<<nWindows, 1>>>(x, y, device_data, device_results);		
		cdprun<<<nWindows, 1>>>(block.x, 0, x, y, device_data, device_results);
		HANDLE_ERROR(hipDeviceSynchronize());
	}
	
      //	fprintf(stderr, "Average kernel time for %d iterations: %lf\n", NITER, NITER/cpu_seconds());


	HANDLE_ERROR(hipMemcpy(host_results, device_results, sizeof(result) * nWindows, hipMemcpyDeviceToHost));

	for(int i = 0; i < nWindows; i++){
		HANDLE_ERROR(hipMemcpy(cigs[i], host_results[i].cigar, sizeof(char) * lenX * 2, hipMemcpyDeviceToHost));
	}

	int overallMax = 0, location = 0, index = 0;
	for(int i = 0; i < nWindows; i++){
		if(overallMax < host_results[i].score){
			overallMax = host_results[i].score;
			location = host_results[i].location;
			index = i;
		}	
	}	
	//char * compressed = compressCigar(cigs[index]);
	printf("Best alignment found at %d :\n", location);
	
	for(int i = 0; i < nWindows; i++){
		free(cigs[i]);
		HANDLE_ERROR(hipFree(host_results[i].cigar));
		HANDLE_ERROR(hipFree(host_data[i].matrix));
	}

	free(host_results);
	free(host_data);
	HANDLE_ERROR(hipFree(x));
	HANDLE_ERROR(hipFree(y));
	HANDLE_ERROR(hipFree(device_data));
	HANDLE_ERROR(hipFree(device_results));
	return 0;
}
